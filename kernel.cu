#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void grayscaleConversion(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * channels;
        unsigned char r = input[rgbOffset];
        unsigned char g = input[rgbOffset + 1];
        unsigned char b = input[rgbOffset + 2];

        unsigned char grayValue = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

        output[grayOffset] = grayValue;
    }
}

int main() {
    const char* inputImagePath = "C:\\Users\\CHARAN\\Downloads\\gr.jpg";
    const char* outputImagePath = "output_image.jpg";

    int width, height, channels;
    unsigned char* inputImage = stbi_load(inputImagePath, &width, &height, &channels, 0);

    if (inputImage == nullptr) {
        std::cerr << "Error loading image: " << inputImagePath << std::endl;
        return -1;
    }

    int imageSize = width * height * channels;

    // Allocate memory on GPU
    unsigned char* d_inputImage;
    unsigned char* d_outputImage;
    hipMalloc((void**)&d_inputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void**)&d_outputImage, width * height * sizeof(unsigned char));

    // Copy input image data from CPU to GPU
    hipMemcpy(d_inputImage, inputImage, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define CUDA grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Launch CUDA kernel for grayscale conversion
    grayscaleConversion << <gridDim, blockDim >> > (d_inputImage, d_outputImage, width, height, channels);

    // Copy output image data from GPU to CPU
    unsigned char* outputImage = new unsigned char[width * height];
    hipMemcpy(outputImage, d_outputImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save grayscale image
    stbi_write_jpg(outputImagePath, width, height, 1, outputImage, 100);

    // Cleanup
    stbi_image_free(inputImage);
    delete[] outputImage;
    hipFree(d_inputImage);
    hipFree(d_outputImage);

    return 0;
}
